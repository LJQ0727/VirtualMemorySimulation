#include "hip/hip_runtime.h"
﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ __managed__ u32 access_trace = 0;  // For LRU implementation.

__device__ void init_invert_page_table(VirtualMemory *vm) {

  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
    vm->invert_page_table[i] = 0x80000000; // invalid := MSB is 1
    vm->invert_page_table[i + vm->PAGE_ENTRIES] = i;
  }
}

__device__ void init_swap_table(VirtualMemory *vm) {

  for (int i = 0; i < vm->SWAP_PAGE_ENTRIES; i++) {
    vm->swap_table[i] = 0x80000000; // invalid := MSB is 1
  }
}

// Initialize the attributes of the VirtualMemory struct
// Initialize the page table by calling `init_invert_page_table`
__device__ void vm_init(VirtualMemory *vm, uchar *buffer, uchar *storage,
                        u32 *invert_page_table, int *pagefault_num_ptr,
                        int PAGESIZE, int INVERT_PAGE_TABLE_SIZE,
                        int PHYSICAL_MEM_SIZE, int STORAGE_SIZE,
                        int PAGE_ENTRIES, u32 *swap_table) {
  // init variables
  vm->buffer = buffer;
  vm->storage = storage;
  vm->invert_page_table = invert_page_table;
  vm->pagefault_num_ptr = pagefault_num_ptr;

  // init constants
  vm->PAGESIZE = PAGESIZE;
  vm->INVERT_PAGE_TABLE_SIZE = INVERT_PAGE_TABLE_SIZE;
  vm->PHYSICAL_MEM_SIZE = PHYSICAL_MEM_SIZE;
  vm->STORAGE_SIZE = STORAGE_SIZE;
  vm->PAGE_ENTRIES = PAGE_ENTRIES;

  vm->swap_table = swap_table;
  vm->SWAP_PAGE_ENTRIES = STORAGE_SIZE / PAGESIZE;


  // before first vm_write or vm_read
  init_invert_page_table(vm);
  init_swap_table(vm);
}

__device__ uchar vm_read(VirtualMemory *vm, u32 addr) {
  /* Complate vm_read function to read single element from data buffer */
  access_trace++;

  return 123; //TODO
}

__device__ int my_log2(int num) {
  // get the log of 2^n, returning n
  int ret = 0;
  while (num != 1)
  {
    num = num >> 1;
    ret++;
  }
  return ret;
}

__device__ u32 alloc_page() {
  // allocate a page from the physical memory
  // return the page number

}

__device__ u32 swap_page() {

}

// Use the LRU algorithm to find a pointer to an entry in the page table that is least recently used
__device__ int LRU_get(VirtualMemory *vm) {
  int swapped_frame_no = 0;
  u32 least_trace_value = vm->invert_page_table[0+vm->PAGE_ENTRIES];

  for (int i = 1; i < vm->PAGE_ENTRIES; i++)
  {
    if (vm->invert_page_table[i+vm->PAGE_ENTRIES] < least_trace_value)
    {
      swapped_frame_no = i;
      least_trace_value = vm->invert_page_table[i+vm->PAGE_ENTRIES];
    }
    
  }
  return swapped_frame_no;
}

__device__ void LRU_put(VirtualMemory *vm, int frame_no) {
  vm->invert_page_table[frame_no+vm->PAGE_ENTRIES] = access_trace;
}

__device__ void swap(VirtualMemory *vm, int swapped_frame_no, int storage_frame_no) {
  // TODO
}

__device__ void vm_write(VirtualMemory *vm, u32 addr, uchar value) {
  /* Complete vm_write function to write value into data buffer */
  access_trace++;

  // given 32-bit virtual address addr, compute the page number and offset fields
  int offset_bit = my_log2(vm->PAGESIZE); // 5-bit offset in each frame (or page)
  int page_entries_bit = my_log2(vm->PAGE_ENTRIES); // 10-bit page entry

  int page_number = addr >> offset_bit; // This page number has at most 13 bits for our problem
  int offset = addr & ((1 << offset_bit) - 1);

  bool page_is_found = false;
  // in the inverted page table, search for the page number
  // i is the frame number
  for (int i = 0; i < vm->PAGE_ENTRIES; i++)
  {
    // if page is found in the page table
    u32 entry = vm->invert_page_table[i];
    if ((entry & 0x80000000 == 0) && entry & 0x7FFFFFFF == page_number) // if the page number is found
    {
      page_is_found = true;
      // get the frame number
      int frame_number = i;
      // write the value into the buffer
      vm->buffer[frame_number * vm->PAGESIZE + offset] = value;
      LRU_put(vm, frame_number);
      return;
    } 
  } 
  
  if (!page_is_found) {
    // if page is not found in the page table
    vm->pagefault_num_ptr[0]++;

    // check if the primary memory is full
    // if the primary memory is not full, we can directly allocate new page there
    for (int i = 0; i < vm->PAGE_ENTRIES; i++)
    {
      if (vm->invert_page_table[i] & 0x80000000 == 1) {
        // this entry is not used
        // mark it occupied
        vm->invert_page_table[i] = 0;

        // write the page number
        vm->invert_page_table[i] &= page_number;

        // write to destination
        vm->buffer[i + offset] = value;

        vm->invert_page_table[i+vm->PAGE_ENTRIES] = access_trace;
        
        return;
      }
    }
    
    // the main memory is full, we have to swap in an exsting page or swap in a new page

    // first, we determine whether the page just doesn't exist yet.
    // find in the swap table
    for (int i = 0; i < vm->SWAP_PAGE_ENTRIES; i++)
    {
      u32 entry = vm->swap_table[i];
      if ((entry & 0x80000000 == 0) && entry & 0x7FFFFFFF == page_number) // if the page number is found
      {
        page_is_found = true;
        
        // the page in found in the swap storage
        // now we swap with an LRU in the main memory
        int swapped_frame_no = LRU_get(vm);

        // do the swapping

        // write the value into the buffer
        vm->buffer[frame_number * vm->PAGESIZE + offset] = value;
        return;
      } 
    }
    
    




    // if not, allocate a page from the physical memory
    int frame_number = alloc_page();
    // write the value into the buffer
    vm->buffer[frame_number * vm->PAGESIZE + offset] = value;
    // update the page table
    vm->invert_page_table[page_number] = frame_number;
    // update the page fault number
    return;
  }
  
  // map to physical address given the page, do the write, set the valid bit


  u32 frame_number = vm->invert_page_table[page_number]; // each page entry is 32-bit
  if (frame_number & 0x80000000 > 0) {
    // allocate a page from the physical memory, or if it's full we'll have to swap
    // The last ten bits of the physical address will be
    frame_number = alloc_page();

    // unset the MSB, indicating valid address

    frame_number = frame_number & 0x7FFFFFFF;
    

  }


  // write to the physical address


}

__device__ void vm_snapshot(VirtualMemory *vm, uchar *results, int offset,
                            int input_size) {
  /* Complete snapshot function togther with vm_read to load elements from data
   * to result buffer */
}

