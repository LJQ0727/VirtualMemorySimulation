﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ void user_program(VirtualMemory *vm, uchar *input, uchar *results,
                             int input_size) {
  for (int i = 0; i < input_size; i++)
    vm_write(vm, i, input[i]);
  // this will produce 16384 page faults

  // for (int i = input_size - 1; i >= input_size - 8192; i--)
  //   int value = vm_read(vm, i);
  // this is just for testing. this will produce 0 additional page faults, becuase we're reading everything in main memory.

  for (int i = input_size - 1; i >= input_size - 32769; i--)
    int value = vm_read(vm, i);
  // this will produce 19460 - 16384 = 3076 page faults
  

  vm_snapshot(vm, results, 0, input_size);
  // the test1 program in total will produce 35844 page faults in my implementation of Case3.
}

// __device__ void user_program(VirtualMemory *vm, uchar *input, uchar *results,
//   int input_size) {
// // write the data.bin to the VM starting from address 32*1024
// for (int i = 0; i < input_size; i++)
//   vm_write(vm, 32*1024+i, input[i]);
// // write (32KB-32B) data  to the VM starting from 0
// for (int i = 0; i < 32*1023; i++)
//   vm_write(vm, i, input[i+32*1024]);
// // readout VM[32K, 160K] and output to snapshot.bin, which should be the same with data.bin
// vm_snapshot(vm, results, 32*1024, input_size);
// }

// expected page fault num: 9215